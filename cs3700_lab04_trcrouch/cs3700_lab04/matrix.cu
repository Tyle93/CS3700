

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define TILE_WIDTH 2
#define WIDTH 6


__global__ void product (float *d_a, float *d_b, float *d_c, const int n) {
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   float sum = 0;
   if (row < n && col < n) {
      for (int i = 0 ; i<n ; ++i) {
         sum += d_a[row * n + i ] * d_b[i * n + col] ;
      }
      d_c[row * n + col] = sum;
   }
}


__global__ void sum(float *d_a, float *d_b, float *d_d, const int n){
   int col = blockIdx.x * blockDim.x + threadIdx.x ;
   int row = blockIdx.y * blockDim.y + threadIdx.y ;

   if (row < n && col < n) {
      for (int i = 0 ; i<n ; ++i) {
         d_d[row * n + i] += d_a[row * n + i ] + d_b[row * n + i] ;
      }
   }
}



void printMatrix (float m[][WIDTH]) {
   int i, j;
   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j< WIDTH; ++j) {
         printf ("%d\t", (int)m[i][j]);
      }
      printf ("\n");
   }
}


int main () {
   // host matrices
   float host_a[WIDTH][WIDTH],
         host_b[WIDTH][WIDTH],
         host_c[WIDTH][WIDTH],
         host_d[WIDTH][WIDTH];

   // device arrays
   float *device_a, *device_b, *device_c, *device_d;

   int i, j;

   // initialize host matrices using random numbers
   time_t t;
   srand ((unsigned) time(&t));

   for (i = 0; i<WIDTH; ++i) {
      for (j = 0; j<WIDTH; j++) {
         host_a[i][j] = (float) (rand() % 50);
         host_b[i][j] = (float) (rand() % 50);
      }
   }

   printf ("Matrix A:\n");
   printMatrix (host_a);
   printf ("\n");

   printf ("Matrix B:\n");
   printMatrix (host_b);
   printf ("\n");

   size_t deviceSize = WIDTH * WIDTH * sizeof (float);
   hipMalloc ((void **) &device_a, deviceSize);
   hipMalloc ((void **) &device_b, deviceSize);

   hipMemcpy (device_a, host_a, deviceSize, hipMemcpyHostToDevice );
   hipMemcpy (device_b, host_b, deviceSize, hipMemcpyHostToDevice );

   hipMalloc((void **) &device_c, deviceSize) ;
   hipMalloc((void **) &device_d, deviceSize) ;

   dim3 dimBlock (WIDTH, WIDTH);
   dim3 dimGrid (WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH);
   product<<<dimGrid, dimBlock>>> (device_a, device_b, device_c, WIDTH);

   hipMemcpy (host_c, device_c, deviceSize, hipMemcpyDeviceToHost);
   sum<<<dimGrid, dimBlock>>> (device_a,device_b,device_d, WIDTH);
   hipMemcpy(host_d, device_d, deviceSize, hipMemcpyDeviceToHost);

   printf ("A x B: \n");
   printMatrix (host_c);
   printf ("\n");

   printf("A + B: \n");
   printMatrix(host_d); 
   printf ("\n");

   hipFree (device_a); 
   hipFree (device_b); 
   hipFree (device_c); 
   hipFree (device_d); 
   return 0;
}